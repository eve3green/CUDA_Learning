﻿#include <iostream>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#define BLOCK_DIM 2 //субматриця
int M = 5*16, K = 5*16;
int* A = new int[M * K]; // матриці в глобальній памя'ті
int* B = new int[M * K];
int* C = new int[M * K];

using namespace std;

__global__ void matrixAdd(int* A, int* B, int* C, int M, int K) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int index = col * M + row;

    //множення
    if (col < M && row < K) {
        C[row * M + col] = 0;
        for (int k = 0; k < M; k++) {
            C[row * M + col] += A[row * M + k] * B[k * M + col];
        }
    }
}

int main() {

    A = new int[M * K];
    B = new int[M * K];
    C = new int[M * K];

    for (int i = 0; i < M; i++)
        for (int j = 0; j < K; j++) {
            A[i * M + j] = 2;
            B[i * M + j] = 2;
            C[i * M + j] = 0;
        }

    int* dev_a, * dev_b, * dev_c; 

    int size = M * K * sizeof(int); //скільки треба виділити пам'яті

    hipMalloc((void**)&dev_a, size); //виділення пам'яті
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);

    hipMemcpy(dev_a, A, size, hipMemcpyHostToDevice); //Перенос на пам'ять ГПУ
    hipMemcpy(dev_b, B, size, hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM); //число выделенных блоков
    dim3 dimGrid((M + dimBlock.x - 1) / dimBlock.x, (K + dimBlock.y - 1) / dimBlock.y); //размер и размерность сетки
    printf("dimGrid.x = %d, dimGrid.y = %d\n", dimGrid.x, dimGrid.y); //выводится размер сетки

    matrixAdd <<<dimGrid, dimBlock >>> (dev_a, dev_b, dev_c, M, K); //викликається ядро
    //cudaDeviceSynchronize(); 

    hipMemcpy(C, dev_c, size, hipMemcpyDeviceToHost);

    //вывод    результата
    printf("Result Matrix C:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < K; j++) {
            printf("%d   ", C[i]);
        }
        printf("\n");
    }


    hipFree(dev_a); //очистка пам'яті
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}
